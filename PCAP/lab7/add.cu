__global__ void add(int* a, int* b, int *c) {
	*c = *a + *b;
}


#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {

	int a, b, c;
	int *d_a, *d_b, *d_c;

	hipMalloc((void**) &d_a, sizeof(int));
	hipMalloc((void**) &d_b, sizeof(int));
	hipMalloc((void**) &d_c, sizeof(int));

	a = 2; b = 234;

	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	add<<<1, 1>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d + %d = %d\n", a, b, c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}