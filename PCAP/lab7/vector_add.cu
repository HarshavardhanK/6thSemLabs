
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(int* a, int* b, int* c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void blocks(int count) {

	int *a = (int*) malloc(sizeof(int) * count);
	int *b = (int*) malloc(sizeof(int) * count);
	int *c = (int*) malloc(sizeof(int) * count);

	int *d_a;
	int *d_b;
	int *d_c;

	for(int i = 0; i < count; i++) {
		a[i] = i;
		b[i] = i + 2;
	}

	hipMalloc((void **)& d_a, sizeof(int) * count);
	hipMalloc((void **)& d_b, sizeof(int) * count);
	hipMalloc((void **)& d_c, sizeof(int) * count);

	hipMemcpy(d_a, a, count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, count * sizeof(int), hipMemcpyHostToDevice);
	
	vector_add<<<count, 256>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, sizeof(int) * count, hipMemcpyDeviceToHost);

	for(int i = 0; i < count; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}


	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

}

__global__ void thread_vect_add(int* a, int* b, int* c) {
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void threads(int count) {

	int *a = (int*) malloc(sizeof(int) * count);
	int *b = (int*) malloc(sizeof(int) * count);
	int *c = (int*) malloc(sizeof(int) * count);

	int *d_a;
	int *d_b;
	int *d_c;

	for(int i = 0; i < count; i++) {
		a[i] = i;
		b[i] = i + 2;
	}

	hipMalloc((void **)& d_a, sizeof(int) * count);
	hipMalloc((void **)& d_b, sizeof(int) * count);
	hipMalloc((void **)& d_c, sizeof(int) * count);

	hipMemcpy(d_a, a, count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, count * sizeof(int), hipMemcpyHostToDevice);
	
	thread_vect_add<<<1, count>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, sizeof(int) * count, hipMemcpyDeviceToHost);

	for(int i = 0; i < count; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}


	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

}

__global__ void thread_per(int* a, int * b, int *c) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

void thread_per_block(int count) {

	int *a = (int*) malloc(sizeof(int) * count);
	int *b = (int*) malloc(sizeof(int) * count);
	int *c = (int*) malloc(sizeof(int) * count);

	int *d_a;
	int *d_b;
	int *d_c;

	for(int i = 0; i < count; i++) {
		a[i] = i;
		b[i] = i + 2;
	}

	hipMalloc((void **)& d_a, sizeof(int) * count);
	hipMalloc((void **)& d_b, sizeof(int) * count);
	hipMalloc((void **)& d_c, sizeof(int) * count);

	hipMemcpy(d_a, a, count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, count * sizeof(int), hipMemcpyHostToDevice);
	
	thread_per<<<ceil(count / 256.0), 256>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, sizeof(int) * count, hipMemcpyDeviceToHost);

	for(int i = 0; i < count; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

int main(int argc, char** argv) {

	if(argc < 2) {
		printf("Enter count in file args\n");
		return -1;
	}

	int count = atoi(argv[1]);

	thread_per_block(count);

	return 0; 

}