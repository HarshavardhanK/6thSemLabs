

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void thread_per(int* a, int * b, int *c, int* alpha) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = *alpha * a[index] + b[index];
}

void thread_per_block(int count) {

	int *a = (int*) malloc(sizeof(int) * count);
	int *b = (int*) malloc(sizeof(int) * count);
	int *c = (int*) malloc(sizeof(int) * count);

	int alpha;

	printf("Enter alpha: ");
	scanf("%d", &alpha);

	int *d_a;
	int *d_b;
	int *d_c;
	int *d_alpha;

	for(int i = 0; i < count; i++) {
		a[i] = i;
		b[i] = i + 2;
	}

	hipMalloc((void **)& d_a, sizeof(int) * count);
	hipMalloc((void **)& d_b, sizeof(int) * count);
	hipMalloc((void **)& d_c, sizeof(int) * count);
	hipMalloc((void **) &d_alpha, sizeof(int));

	hipMemcpy(d_a, a, count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_alpha, &alpha, sizeof(int), hipMemcpyHostToDevice);
	
	thread_per<<<ceil(count / 256.0), 256>>>(d_a, d_b, d_c, d_alpha);

	hipMemcpy(c, d_c, sizeof(int) * count, hipMemcpyDeviceToHost);

	for(int i = 0; i < count; i++) {
		printf("%d * %d + %d = %d\n", alpha, a[i], b[i], c[i]);
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

int main(int argc, char** argv) {

	if(argc < 2) {
		printf("Enter count in file args\n");
		return -1;
	}

	int count = atoi(argv[1]);

	thread_per_block(count);

	return 0;

}