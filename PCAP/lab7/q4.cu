
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void thread_per(float* a, float * b) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	b[index] = sinf(a[index]);
}

void thread_per_block(int count) {

	float a[] = {0.0, 1.57, 2.57, 3.14};
	float *b = (float*) malloc(sizeof(float) * 4);

	float *d_a;
	float *d_b;

	hipMalloc((void **)& d_a, sizeof(float) * 4);
	hipMalloc((void **)& d_b, sizeof(float) * 4);

	hipMemcpy(d_a, a, count * sizeof(float), hipMemcpyHostToDevice);
	
	thread_per<<<ceil(count / 256.0), 256>>>(d_a, d_b);

	hipMemcpy(b, d_b, sizeof(float) * 4, hipMemcpyDeviceToHost);

	for(int i = 0; i < count; i++) {
		printf("sin(%.2f) = %.2f\n", a[i], b[i]);
	}

	hipFree(d_a);
	hipFree(d_b);
}

int main(int argc, char** argv) {

	if(argc < 2) {
		printf("Enter count in file args\n");
		return -1;
	}

	int count = atoi(argv[1]);

	thread_per_block(count);

	return 0;

}
